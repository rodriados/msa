#include "hip/hip_runtime.h"
/**
 * Multiple Sequence Alignment hybrid neighbor-joining file.
 * @author Rodrigo Siqueira <rodriados@gmail.com>
 * @copyright 2019-2020 Rodrigo Siqueira
 */
#include <limits>
#include <cstdint>
#include <utility>

#include <cuda.cuh>
#include <node.hpp>
#include <oeis.hpp>
#include <utils.hpp>
#include <buffer.hpp>
#include <matrix.hpp>
#include <pairwise.cuh>
#include <exception.hpp>
#include <transform.hpp>
#include <environment.h>

#include <phylogeny/matrix.cuh>
#include <phylogeny/phylogeny.cuh>
#include <phylogeny/algorithm/njoining.cuh>

namespace
{
    using namespace msa;
    using namespace phylogeny;

    /*
     * Algorithm configuration parameters. These values interfere directly into
     * the algorithm's execution, thus, they shall be modified with caution.
     */
    enum : size_t { reduce_factor = 2 };

    /**
     * The algorithm's distance type. 
     * @since 0.1.1
     */
    using distance_type = pairwise::score;

    /**
     * The type for mapping an OTU to its coordinates on the matrix.
     * @since 0.1.1
     */
    using map_type = buffer<oturef>;

    /**
     * Defines a cache for the matrix's columns and row sums.
     * @since 0.1.1
     */
    using cache_type = buffer<distance_type>;

    /**
     * The point type required by the algorithm's matrices.
     * @since 0.1.1
     */
    using pair_type = typename msa::matrix<distance_type>::point_type;

    /**
     * The neighbor-joining algorithm's star tree data structures.
     * @tparam T The star tree's matrix spatial transformation type.
     * @since 0.1.1
     */
    template <typename T>
    struct startree
    {
        phylogeny::matrix<true, T> matrix;  /// The algorithm's distance matrix.
        cache_type cache;                   /// The cache of lines and columns total sums.
        map_type map;                       /// The OTU references map to matrix indeces.
        size_t count;                       /// The number of OTUs yet to be joined.
    };

    /**
     * The reduceable interface. This interface's implementations shall be forced
     * to have a method to join two elements at the given offsets together.
     * @tparam T The type of elements to be reduced.
     * @since 0.1.1
     */
    template <typename T>
    struct reduceable
    {
        __device__ static inline void join(volatile T *, size_t, size_t);
    };

    /**
     * Calculates the highest number which is a power of 2 and is smaller than or
     * equal to the given input.
     * @param x The target input number.
     * @return The resulting power of 2.
     */
    static inline uint32_t floor_power2(uint32_t x) noexcept
    {
      #if !defined(__msa_compiler_gnuc)
        x |= x >> 1;
        x |= x >> 2;
        x |= x >> 4;
        x |= x >> 8;
        x |= x >> 16;
        x |= x >> 32;
        return x ^ (x >> 1);
      #else
        return 0x80000000 >> __builtin_clz(x);
      #endif
    }

    /**
     * Performs a reduce inside a single warp.
     * @tparam N The number of total elements being reduced.
     * @tparam T The type of elements being reduced.
     * @param data The data array being reduced into a single value.
     * @param offset The current thread's offset to process.
     */
    template <typename R, typename T>
    __device__ inline void reduce(volatile T *data, size_t count, size_t offset)
    {
        static_assert(std::is_base_of<reduceable<T>, R>::value, "invalid reduceable type");

        switch(count) {
            case 1024: if(offset < 512) { R::join(data, offset, offset + 512); } __syncthreads();
            case  512: if(offset < 256) { R::join(data, offset, offset + 256); } __syncthreads();
            case  256: if(offset < 128) { R::join(data, offset, offset + 128); } __syncthreads();
            case  128: if(offset <  64) { R::join(data, offset, offset +  64); } __syncthreads();
            case   64: if(offset <  32) { R::join(data, offset, offset +  32); }
            case   32: if(offset <  16) { R::join(data, offset, offset +  16); }
            case   16: if(offset <   8) { R::join(data, offset, offset +   8); }
            case    8: if(offset <   4) { R::join(data, offset, offset +   4); }
            case    4: if(offset <   2) { R::join(data, offset, offset +   2); }
            case    2: if(offset <   1) { R::join(data, offset, offset +   1); }
        }
    }

    /**
     * Fills the star tree's distances sum cache on device memory.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param star The star tree's object instance.
     */
    template <typename T>
    __global__ void fill_cache(startree<T> star)
    {
        extern __shared__ distance_type sums[];

        // Implements the reduction operation for filling the star tree's matrix
        // columns and lines cache. As we are interested on building a cache with
        // the total sum of the matrix's lines and columns, this operation shall
        // simply accumulate by summing two data offsets into one.
        using sum = struct : reduceable<distance_type> {
            __device__ static inline void join(volatile distance_type *data, size_t dest, size_t src) {
                data[dest] += data[src];
            }
        };

        // For each of the star tree's matrix's columns and lines, we must iterate
        // over their elements and sum them all together in order to fill our cache.
        for(int32_t i = blockIdx.x; i < star.count; i += gridDim.x) {
            sums[threadIdx.x] = 0;

            // As we cannot spawn a thread to every single element of our cache
            // or distance matrix, we must "manually" sum every exceeding element
            // so that these elements are included when we reduce our shared array.
            for(int32_t j = threadIdx.x; j < star.count; j += blockDim.x)
                sums[threadIdx.x] += star.matrix[{i, j}];

            __syncthreads();

            // Now that out shared array is ready to be reduced, and all exceeding
            // elements have been summed, we can perform our reduce operation.
            reduce<sum>(sums, blockDim.x, threadIdx.x);

            if(threadIdx.x == 0)
                star.cache[i] = sums[0];
        }
    }

    /**
     * Builds a cache for the sum of all elements from a matrix's columns and rows.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param star The algorithm's star tree to initialize the sum cache of.
     */
    template <typename T>
    static void cache_init(startree<T>& star)
    {
        using namespace cuda::device;
        const auto height = (int32_t) star.matrix.dimension()[0];
        const auto width  = (int32_t) star.matrix.dimension()[1];

        // The number of threads spawned by each block to initialize our cache will
        // be a power of 2 roughly equal to half the width of our matrix. We force
        // such specific number in order to take the most out of our reduce kernel.
        const auto blocks  = max_blocks(height);
        const auto threads = floor_power2(max_threads(width / reduce_factor));

        fill_cache<<<blocks, threads, sizeof(distance_type) * threads>>>(star);
    }

    /**
     * Initialize a new star tree, and builds all data structures needed for a fast
     * neighbor-joining execution.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param matrix The pairwise module's distance matrix.
     * @param count The total number of OTUs to be aligned.
     * @return The initialized star tree.
     */
    template <typename T>
    static auto initialize(const pairwise::distance_matrix& matrix, size_t count) -> startree<T>
    {
        startree<T> star;
        auto hmat = phylogeny::matrix<false, T> {matrix};

        star.count = count;
        star.matrix = hmat.to_device();
        star.map = map_type::make(count);

        onlyslaves star.cache = cache_type::make(cuda::allocator::device, star.count);
        onlyslaves cache_init(star);

        for(size_t i = 0; i < count; ++i)
            star.map[i] = (otu) i;

        return star;
    }

    /**
     * Calculates the Q-value for the given OTU pair.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param star The OTUs' star tree data structures.
     * @param pair The target pair to get the Q-value of.
     * @return The given pair's Q-value.
     */
    template <typename T>
    __device__ inline distance_type q_transform(const startree<T>& star, const pair_type& pair)
    {
        return (star.count - 2) * star.matrix[pair] - star.cache[pair.x] - star.cache[pair.y];
    }

    /**
     * Raises a candidate OTU pair into the local best joinable OTU pair.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param star The OTUs' star tree data structures.
     * @param chosen The chosen candidate as the local best OTU pair.
     * @return The fully-joinable OTU pair.
     */
    template <typename T>
    __device__ inline njoining::joinable raise_candidate(
            const startree<T>& star
        ,   const njoining::candidate& chosen
        )
    {
        const pair_type pair = {chosen.ref[0], chosen.ref[1]};
        const auto pairsum = star.cache[pair.x] - star.cache[pair.y];

        const distance_type dx = (.5 * star.matrix[pair]) + (pairsum / (2 * (star.count - 2)));
        const distance_type dy = star.matrix[pair] - dx;

        return {chosen, dx, dy};
    }

    /**
     * Finds the local best OTU candidates to be joined.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param result The kernel's final result buffer.
     * @param star The OTUs' star tree data structures.
     * @param partition The local range at which a candidate must be found.
     */
    template <typename T>
    __global__ void find_candidates(
            buffer<njoining::joinable> result
        ,   const startree<T> star
        ,   const range<size_t> partition
        )
    {
        extern __shared__ njoining::candidate list[];
        new (&list[threadIdx.x]) njoining::candidate {};

        // Implements a reduction operation for finding the local best OTU pair
        // to be joined next. As we apply the Q-transformation for every pair on
        // the distance matrix, we must find the one with the smallest Q-value.
        using min = struct : reduceable<njoining::candidate> {
            __device__ static inline void join(volatile njoining::candidate *data, size_t dest, size_t src) {
                if(data[src].distance < data[dest].distance) data[dest] = data[src];
            }
        };

        // As we cannot spawn a thread for every single pair we must calculate,
        // we have to process the excess before reducing our shared array.
        for(size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < partition.total; i += gridDim.x * blockDim.x) {
            const size_t x = oeis::a002024(partition.offset + i + 1);
            const size_t y = (partition.offset + i) - utils::nchoose(x);
            const auto distance = q_transform(star, {x, y});

            if(distance < list[threadIdx.x].distance)
                list[threadIdx.x] = njoining::candidate {x, y, distance};
        }

        __syncthreads();

        // Reduces the shared list of candidates to find the absolute local best
        // on the current device. The list has already been reduced to a smaller
        // amount due to the operation performed above.
        reduce<min>(list, blockDim.x, threadIdx.x);

        if(threadIdx.x == 0)
            result[blockIdx.x] = raise_candidate(star, list[0]);
    }

    /**
     * Finds the best joinable pair on the given partition.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param star The OTUs' star tree data structures.
     * @param partition The local range at which a candidate must be found.
     * @return The best joinable pair candidate found on the given partition.
     */
    template <typename T>
    static njoining::joinable pick_joinable(const startree<T>& star, const range<size_t>& partition)
    {
        using namespace cuda::device;

        // The number of threads spawned by each block to find the local best joinable
        // OTU will be a power of 2 roughly equal to half the partition size. Also,
        // we only spawn new blocks if all of its threads will be used.
        const size_t total = partition.total / reduce_factor;
        const auto threads = floor_power2(max_threads(total));
        const auto blocks  = max_blocks(total / threads);

        auto result = buffer<njoining::joinable>::make(blocks);
        auto chosen = buffer<njoining::joinable>::make(cuda::allocator::device, blocks);
        size_t smallest = 0;

        find_candidates<<<blocks, threads, sizeof(njoining::candidate) * threads>>>(chosen, star, partition);
        cuda::memory::copy(result.raw(), chosen.raw(), blocks);

        // Now that we reduced the total number of candidates, we can finally apply
        // a small reduction to find the absolute best on this node's partition.
        for(size_t i = 1; i < blocks; ++i)
            if(result[i].distance < result[smallest].distance)
                smallest = i;

        return result[smallest];
    }

    /**
     * Swaps the given pair of OTUs and removes one of them from the star tree.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param star The OTU's star tree data structures.
     * @param keep The OTU to be swapped but kept in the star tree.
     * @param remove The OTU to be swapped and removed from the star tree.
     */
    template <typename T>
    static void swap_remove(startree<T>& star, oturef keep, oturef remove)
    {
        onlyslaves {
            star.matrix.swap(keep, remove);
            star.matrix.remove(remove);
        }

        ptrdiff_t shift = (remove == 0);
        utils::swap(star.map[keep], star.map[remove]);
        star.map = map_type {star.map.offset(shift), star.map.size() - 1};
        onlyslaves star.cache = cache_type {star.cache.offset(shift), star.cache.size() - 1};
    }

    /**
     * Updates the star tree's cache structures by removing an OTU.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param star The OTU's star tree data structures.
     * @param x The OTU to be removed from the star tree's caches and matrix.
     */
    template <typename T>
    static void update_cache(startree<T>& star, oturef x)
    {
        if(std::is_same<transform::symmetric, T>::value) {
            swap_remove(star, x, 0);
        } else {
            swap_remove(star, x, star.count - 1);
        }
    }

    /**
     * Rebuils the star tree's matrix and cache while joining neighboring OTUs.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param star The OTU's star tree data structures.
     * @param pair The pair of OTUs being currently joined.
     */
    template <typename T>
    __global__ void rebuild(startree<T> star, const pair_type pair)
    {
        __shared__ distance_type new_sum;
        
        if(threadIdx.x == 0)
            new_sum = 0;

        __syncthreads();

        // Calculate the distances from the new OTU, being created from the join
        // of the two given OTUs, to all other unmodified OTUs.
        for(size_t i = threadIdx.x; i < star.count; i += blockDim.x) {
            const auto previous = star.matrix[{i, pair.x}] + star.matrix[{i, pair.y}];
            const auto current = .5 * (previous - star.matrix[pair]);

            star.matrix[{i, pair.x}] = star.matrix[{pair.x, i}] = current;
            star.cache[i] += current - previous;

            atomicAdd(&new_sum, current);
        }

        __syncthreads();

        // Updates the new OTU's cache to reflect its total distances sum and removes
        // one of the older OTUs from the star tree's sum cache.
        if(threadIdx.x == 0) {
            std::is_same<transform::symmetric, T>::value
                ? utils::swap(star.cache[pair.y], star.cache[0])
                : utils::swap(star.cache[pair.y], star.cache[star.count - 1]);
            star.cache[pair.x] = new_sum;
        }
    }

    /**
     * Joins an OTU pair into a new parent OTU.
     * @tparam T The star tree's matrix spatial transformation type.
     * @param phylotree The phylogenetic tree being constructed.
     * @param parent The parent OTU into which the pair will be joined.
     * @param star The OTU's star tree data structures.
     * @param join The OTU pair to join.
     */
    template <typename T>
    static void join_pair(tree& phylotree, oturef parent, startree<T>& star, const njoining::joinable& join)
    {
        using namespace cuda::device;

        const auto x = join.ref[0];
        const auto y = join.ref[1];

        // As updating the star tree is a computationally expensive task, we optimize
        // it by reusing one of the joined OTU's column and row on the matrix to
        // store the new OTU's distances.
        phylotree.join(parent, {star.map[x], join.delta[0]}, {star.map[y], join.delta[1]});

        // Let's calculate the distances between the OTU being created and the others
        // which have not been affected by the current joining operation.
        onlyslaves rebuild<<<1, max_threads(star.count)>>>(star, {x, y});

        // Finally, let's take advantage from our data structures' layouts and always remove
        // the cheapest column from our star tree's distance matrix.
        star.map[x] = parent;

        update_cache(star, y);
        --star.count;
    }

    /**
     * The hybrid neighbor-joining algorithm object. This algorithm uses hybrid
     * parallelism to run the Neighbor-Joining algorithm.
     * @tparam T The matrix spatial transformation to use within the algorithm.
     * @since 0.1.1
     */
    template <typename T>
    struct hybrid : public njoining::algorithm
    {
        /**
         * Builds the pseudo-phylogenetic tree from the given distance matrix.
         * @param matrix The distance matrix to build tree from.
         * @param count The total number of leaves in tree.
         * @return The calculated phylogenetic tree.
         */
        auto build_tree(startree<T>& star) const -> tree
        {
            oturef parent = (otu) star.count;
            auto phylotree = tree::make(star.count);

            // We must keep joining OTU pairs until there are only three OTUs left
            // in our star tree, so all the other OTUs have been joined.
            while(star.count > 2) {
                range<size_t> partition;
                njoining::joinable vote;

                const size_t total = utils::nchoose(star.count);

                // Let's split the total amount of work to be done between our compute
                // nodes. Each node must pick its local best joinable candidate.
                #if !defined(__msa_runtime_cython)
                    onlyslaves partition = utils::partition(total, node::count - 1, node::rank - 1);
                #else
                    partition = range<size_t> {0, total};
                #endif

                // After finding each compute node's local best joinable candidate,
                // we must gather the votes and find the best one globally.
                onlyslaves vote = pick_joinable(star, partition);
                vote = this->reduce(vote);

                // At last, we join the selected pair, rebuild our distance matrix
                // with the newly created OTU, recalculate our sum cache with the
                // new OTU and update our OTU map to reflect the changes.
                join_pair(phylotree, parent++, star, vote);
            }

            return phylotree;
        }

        /**
         * Executes the sequential neighbor-joining algorithm for the phylogeny
         * step. This method is responsible for coordinating the execution.
         * @return The module's result value.
         */
        auto run(const context& ctx) const -> tree override
        {
            if (ctx.total < 2)
                return tree {};

            auto star = initialize<T>(ctx.matrix, ctx.total);
            auto result = build_tree(star);

            return result;
        }
    };
}

namespace msa
{
    /**
     * Instantiates a new hybrid neighbor-joining instance using a simple matrix.
     * @return The new algorithm instance.
     */
    extern auto phylogeny::njoining::hybrid_linear() -> phylogeny::algorithm *
    {
        return new ::hybrid<transform::linear<2>>;
    }

    /**
     * Instantiates a new hybrid neighbor-joining instance using a symmatrix.
     * @return The new algorithm instance.
     */
    extern auto phylogeny::njoining::hybrid_symmetric() -> phylogeny::algorithm *
    {
        return new ::hybrid<transform::symmetric>;
    }
}
