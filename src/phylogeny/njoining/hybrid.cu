
#include <hip/hip_runtime.h>
/**
 * Multiple Sequence Alignment hybrid neighbor-joining file.
 * @author Rodrigo Siqueira <rodriados@gmail.com>
 * @copyright 2019 Rodrigo Siqueira
 */
/*#include "msa.hpp"
#include "cuda.cuh"
#include "pointer.hpp"
#include "pairwise.cuh"
#include "cartesian.hpp"
#include "exception.hpp"

#include "phylogeny/tree.cuh"
#include "phylogeny/matrix.cuh"
#include "phylogeny/njoining.cuh"
#include "phylogeny/phylogeny.cuh"

using namespace phylogeny;

namespace
{
    JoinablePair findLocalPair(const ShrinkableMatrix<Score>& mat)
    {
        
    }*/

    /**
     * The hybrid neighbor-joining algorithm object. This algorithm uses
     * hybrid parallelism to run the Neighbor-Joining algorithm.
     * @since 0.1.1
     */
    //struct Hybrid : public NJoining
    //{
        /**
         * Executes the hybrid neighbor-joining algorithm for the phylogeny step.
         * This method is responsible for distributing and gathering workload
         * from different cluster nodes.
         * @param config The module's configuration.
         * @return The module's result value.
         */
        /*Tree run(const Configuration& config) override
        {
            auto mat = TriangularMatrix<Score>::fromPairwise(config.pw).toDevice();
            this->tree = Tree {config.pw.getCount()};

            //auto pair = this->synchronize(selectMinimum(mat));

            while(mat.getCount() > 3) {
                //this->tree.join(pair.id[0], pair.id[1]);
                //pair = rebuildAndNext(mat, pair);
            }

            return this->tree;
        }
    };
};*/

/**
 * Instantiates a new hybrid neighbor-joining instance.
 * @return The new algorithm instance.
 */
/*extern Algorithm *njoining::hybrid()
{
    return new Hybrid;
}*/