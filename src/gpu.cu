/** 
 * Multiple Sequence Alignment GPU tools file.
 * @author Rodrigo Siqueira <rodriados@gmail.com>
 * @copyright 2018 Rodrigo Siqueira
 */
#include <hip/hip_runtime.h>

#include "msa.hpp"
#include "gpu.hpp"
#include "interface.hpp"

extern cli::Data cli_data;
extern mpi::Data mpi_data;

/**
 * Informs the number of GPU devices connected.
 * @return The number of GPU devices found.
 */
int gpu::count()
{
    int count = 0;

    __cudacheck(hipGetDeviceCount(&count));
    return cli_data.multigpu ? count : 1;
}

/**
 * Checks whether at least one GPU device is connected.
 * @return Is there any GPU device connected?
 */
bool gpu::check()
{
    return count() > 0;
}

/**
 * Checks whether more than one GPU devices are connected.
 * @return Are there more than one GPU devices connected?
 */
bool gpu::multi()
{
    return cli_data.multigpu && count() > 1;
}

/**
 * Assigns a GPU device according to the process rank.
 * @return Assigned GPU identifier.
 */
int gpu::assign()
{
    return cli_data.multigpu
        ? (mpi_data.rank - 1) % count()
        : 0;
}
