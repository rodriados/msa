/** 
 * Multiple Sequence Alignment CUDA tools file.
 * @author Rodrigo Siqueira <rodriados@gmail.com>
 * @copyright 2018-2019 Rodrigo Siqueira
 */
#include <hip/hip_runtime.h>
#include <string>

#include <cuda.cuh>

/**
 * Obtain a brief textual explanation for a specified kind of CUDA Runtime API
 * status or error code.
 * @param code The error code to be described.
 * @return The error description.
 */
std::string cuda::status::describe(cuda::status_code code) noexcept
{
    return hipGetErrorString(static_cast<hipError_t>(code));
}

/**
 * Gets the number of devices available.
 * @return The number of devices or runtime error.
 */
auto cuda::device::count() -> size_t
{
    int devices;
    cuda::check(hipGetDeviceCount(&devices));
    return static_cast<size_t>(devices);
}

/**
 * Gets the current device id.
 * @return The device id or runtime error.
 */
auto cuda::device::current() -> cuda::device::id
{
    int device;
    cuda::check(hipGetDevice(&device));
    return device;
}

/**
 * Sets the current device to given id.
 * @param device The device to be used.
 */
auto cuda::device::select(const cuda::device::id& device) -> void
{
    cuda::check(hipSetDevice(device));
}

/**
 * Retrieves information and properties about the chosen device.
 * @param device The device of which properties will be retrieved.
 * @return The device properties.
 */
auto cuda::device::properties(const cuda::device::id& device) -> cuda::device::props
{
    cuda::device::props props;
    cuda::check(hipGetDeviceProperties(&props, device));
    return props;
}
