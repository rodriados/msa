/** 
 * Multiple Sequence Alignment CUDA tools file.
 * @author Rodrigo Siqueira <rodriados@gmail.com>
 * @copyright 2018-2019 Rodrigo Siqueira
 */
#include <hip/hip_runtime.h>
#include <string>

#include <cuda.cuh>
#include <allocator.hpp>

namespace msa
{
    /**
     * The allocator instance for reserving and managing pointers of memory regions
     * allocated in device's global memory space.
     * @since 0.1.1
     */
    allocator cuda::allocator::device = {
        [](void **ptr, size_t size, size_t n) { cuda::check(hipMalloc(ptr, size * n)); }
    ,   [](void *ptr) { cuda::check(hipFree(ptr)); }
    };

    /**
     * The allocator instance for reserving and managing pointers to pinned host-side
     * memory regions. Pinned memory is unpaginated and thus can be accessed faster
     * by the device's internal instructions.
     * @since 0.1.1
     */
    allocator cuda::allocator::pinned = {
        [](void **ptr, size_t size, size_t n) { cuda::check(hipHostMalloc(ptr, size * n)); }
    ,   [](void *ptr) { cuda::check(hipHostFree(ptr)); }
    };

    /**
     * Obtain a brief textual explanation for a specified kind of CUDA Runtime 
     * API status or error code.
     * @param code The error code to be described.
     * @return The error description.
     */
    std::string cuda::status::describe(cuda::status_code code) noexcept
    {
        return hipGetErrorString(static_cast<hipError_t>(code));
    }

    /**
     * Gets the number of devices available.
     * @return The number of devices or runtime error.
     */
    auto cuda::device::count() -> size_t
    {
        int devices;
        cuda::check(hipGetDeviceCount(&devices));
        return static_cast<size_t>(devices);
    }

    /**
     * Gets the current device id.
     * @return The device id or runtime error.
     */
    auto cuda::device::current() -> cuda::device::id
    {
        int device;
        cuda::check(hipGetDevice(&device));
        return device;
    }

    /**
     * Sets the current device to given id.
     * @param device The device to be used.
     */
    auto cuda::device::select(const cuda::device::id& device) -> void
    {
        cuda::check(hipSetDevice(device));
    }

    /**
     * Retrieves information and properties about the chosen device.
     * @param device The device of which properties will be retrieved.
     * @return The device properties.
     */
    auto cuda::device::properties(const cuda::device::id& device) -> cuda::device::props
    {
        cuda::device::props props;
        cuda::check(hipGetDeviceProperties(&props, device));
        return props;
    }
}